#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define DATA float
#define BOOL int
#define MAX_ERR 1e-5

//Grid features

#define OPTIMUM_BLOCK_NUM 4 
#define BLOCK_SIDE	16 

#define OPTIMUM_BLOCK_NUM_FIRST_LAYER 2
#define BLOCK_SIDE_FIRST_LAYER 32

/*Struct Grid Settings*/

typedef struct grid_settings {
	unsigned int grid[3];
	unsigned int block[3];
}grid_settings;

grid_settings gs = { { OPTIMUM_BLOCK_NUM_FIRST_LAYER, OPTIMUM_BLOCK_NUM, OPTIMUM_BLOCK_NUM },{ BLOCK_SIDE_FIRST_LAYER,BLOCK_SIDE,BLOCK_SIDE } };

//Network features

#define NEURO_INPUT 784 //#neurons of input layer
#define NEURO_H_0	56	//#neurons of first hidden layer
#define NEURO_H_1	28	//#neurons of second hidden layer
#define NEURO_OUTPUT 10 //#neurons of output layer
#define TOTAL_PATT	60000 //#total patterns
#define NUM_HIDDEN 2 //#hidden layers
#define TOTAL_LAYER 4 //#of layers

//Streams Settings
#define NSTREAMS 3
#define STREAMSIZE TOTAL_PATT/NSTREAMS

//Texture reference (FOR TARGET MATRIX)
texture<DATA, 2, hipReadModeElementType> texreference_target;

/*UTILITIES*/

static void HandleCuda(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_CUDA( err ) (HandleCuda( err, __FILE__, __LINE__ ))

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventCreate(start));
	HANDLE_CUDA(hipEventCreate(stop));
	HANDLE_CUDA(hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventRecord(*stop, 0));
	HANDLE_CUDA(hipEventSynchronize(*stop));
	float time = 0.0f;
	HANDLE_CUDA(hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f milliseconds\n", time);
	HANDLE_CUDA(hipEventDestroy(*start));
	HANDLE_CUDA(hipEventDestroy(*stop));
}

/*DEVICE*/

__device__ void MMMulDevPartial(DATA *, DATA *, DATA *, DATA *, unsigned int, unsigned int, unsigned int);
__global__ void MMMulDev(DATA *, DATA *, DATA *, DATA *, unsigned int, unsigned int, unsigned int);

/*HOST*/
void feedforward(DATA *, DATA **, DATA **, DATA **, DATA **, DATA **, int *, int);
void HOST_feedforward(DATA *, DATA **, DATA **, DATA **, int *);
void printMat(DATA *, int, int);
void MMMulHost(DATA *, DATA *, DATA *, DATA *, int, int, int);
BOOL matsAreEquals(DATA *, DATA *, int, int);

/*HOST ALLOCATION AND INITIALIZATION*/

void HOST_alloc_init(DATA**, DATA**, DATA**, int*);

/*DEVICE ALLOCATION*/

void CUDA_alloc(DATA**, DATA**, DATA**, int*);

/*HOST DEALLOCATION FUNCTIONS*/

void HOST_dealloc(DATA*, DATA*, DATA**, DATA**, DATA**);

/*DEVICE DEALLOCATION FUNCTIONS*/

void CUDA_dealloc(DATA**, DATA**, DATA**);

/*----------------------------------------------------------------------MAIN---------------------------------------------------------------------------*/

int main(void) {

	DATA *INPUT_MAT, **H2H_MAT, **W_MAT, **BIAS_MAT, **DEV_H2H_MAT, **DEV_W_MAT, **DEV_BIAS_MAT;
	/*---------------------------CUDA ARRAY FOR TEXTURE-----------------------------*/
	hipArray* DEV_TARGET_CUDA;
	hipChannelFormatDesc channel;
	/*---------------------------END CUDA ARRAY FOR TEXTURE-------------------------*/
	DATA *TARGET;//HOST
	DATA *OUTPUT_MAT;//HOST UTILITY DATA
	int *nupl = (int*)malloc(TOTAL_LAYER * sizeof(int));//Number of neurons per layer

	/*questa parte bisogner� renderla dinamica. In seguito bisogner� accedere ai files.*/
	nupl[0] = NEURO_INPUT;
	nupl[1] = NEURO_H_0;
	nupl[2] = NEURO_H_1;
	nupl[TOTAL_LAYER - 1] = NEURO_OUTPUT;

	/*----------------TARGET AND INPUT (PINNED) ALLOCATION AND INITIALIZATION (HOST)-----------------*/
	
	TARGET = (DATA*)malloc(NEURO_OUTPUT*TOTAL_PATT * sizeof(DATA)); //TARGET OF THE PATTERNS
	
	for (int i = 0; i < TOTAL_PATT; i++) {
		for (int j = 0; j < NEURO_OUTPUT; j++) {
			TARGET[i*NEURO_OUTPUT + j] = (DATA)rand() / (DATA)RAND_MAX;
		}
	}

	//INPUT PINNED
	HANDLE_CUDA(hipHostAlloc(&INPUT_MAT, nupl[0] * TOTAL_PATT * sizeof(DATA), 0));

	for (int i = 0; i < TOTAL_PATT; i++) {
		for (int j = 0; j < NEURO_INPUT; j++) {
			INPUT_MAT[i*NEURO_INPUT + j] = (DATA)rand() / (DATA)RAND_MAX;
		}
	}

	/*host memory allocation and initialization*/
	H2H_MAT = (DATA**)malloc((TOTAL_LAYER - 1) * sizeof(DATA*));
	W_MAT = (DATA**)malloc((TOTAL_LAYER - 1) * sizeof(DATA*));
	BIAS_MAT = (DATA**)malloc((TOTAL_LAYER - 1) * sizeof(DATA*));
	HOST_alloc_init(H2H_MAT, W_MAT, BIAS_MAT, nupl);

	OUTPUT_MAT = (DATA*)malloc(NEURO_OUTPUT*TOTAL_PATT * sizeof(DATA));

	/*device memory allocation*/
	DEV_H2H_MAT = (DATA**)malloc(TOTAL_LAYER * sizeof(DATA*));
	DEV_W_MAT = (DATA**)malloc((TOTAL_LAYER - 1) * sizeof(DATA*));
	DEV_BIAS_MAT = (DATA**)malloc((TOTAL_LAYER - 1) * sizeof(DATA*));
	CUDA_alloc(DEV_H2H_MAT, DEV_W_MAT, DEV_BIAS_MAT, nupl);

	/*device allocation of Target cuda array, memcpy from host Target matrix and texture reference binding*/
	channel = hipCreateChannelDesc<DATA>();
	HANDLE_CUDA(hipMallocArray(&DEV_TARGET_CUDA, &channel, NEURO_OUTPUT, TOTAL_PATT));
	HANDLE_CUDA(hipMemcpyToArray(DEV_TARGET_CUDA,0,0,TARGET,NEURO_OUTPUT*TOTAL_PATT*sizeof(DATA),hipMemcpyHostToDevice));

	texreference_target.filterMode = hipFilterModePoint;
	texreference_target.addressMode[0] = hipAddressModeWrap;
	texreference_target.addressMode[1] = hipAddressModeClamp;

	HANDLE_CUDA(hipBindTextureToArray(texreference_target, DEV_TARGET_CUDA)); //Texture reference binding

	/*-----------------------------------FEEDFORWARD-------------------------------------------*/

	hipEvent_t start, stop;

	startTimer(&start, &stop);
	feedforward(INPUT_MAT, W_MAT, BIAS_MAT, DEV_H2H_MAT, DEV_W_MAT, DEV_BIAS_MAT, nupl, TOTAL_LAYER);
	stopAndPrint(&start, &stop);
	hipDeviceSynchronize(); //necessary

	/*-----------------------------END---FEEDFORWARD-------------------------------------------*/

	//Host dealloc
	free(nupl);
	HOST_dealloc(INPUT_MAT, TARGET, H2H_MAT, W_MAT, BIAS_MAT);
	//Unbinding texture
	hipUnbindTexture(texreference_target);
	//Cuda dealloc
	CUDA_dealloc(DEV_H2H_MAT, DEV_W_MAT, DEV_BIAS_MAT);
	free(DEV_H2H_MAT);
	free(DEV_W_MAT);
	free(DEV_BIAS_MAT);
	hipFreeArray(DEV_TARGET_CUDA); //Free cuda array
	//Free host utility data
	free(OUTPUT_MAT);

	return 0;
}


/*---------------------------------------------------------------------KERNEL--------------------------------------------------------------------------*/

/*DEVICE*/

/* h2h � il puntatore alla porzione dell'h2h globale da considerare in questa fase
(ad ogni passo il kernel che invoca questo device incrementa il puntatore h2h
in modo proporzionale al patt_per_step (e similmente h2h_dest) (vedi sotto))*/

__device__ void MMMulDevPartial(DATA *h2h, DATA *w, DATA *biases, DATA * h2h_dest, unsigned int row_w, unsigned int col_w, unsigned int num_pattern) {

	int tx = threadIdx.x, ty = threadIdx.y;
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	const int block_dim = blockDim.x; // assumiamo che i blocchi siano quadrati
	int dest_x = block_x*block_dim + tx;
	int dest_y = block_y*block_dim + ty;

	int w_x = block_x*block_dim; // start block in w
	int h2h_y = block_y*block_dim*row_w; // start block in h2h

	int end_h2h = h2h_y + row_w - 1; // last block position in h2h

	int step_w = block_dim*col_w;
	int step_h2h = block_dim;
	int min;

	DATA partial = 0.0f;
	int block_r_border = 0; // contatore che indica in che iterazione dei blocchi ci troviamo
	int current_inc;

	for (int wid = w_x, h2h_id = h2h_y; h2h_id <= end_h2h; wid += step_w, h2h_id += step_h2h) {

		block_r_border += block_dim;

		//__shared__ DATA shared_w[BLOCK_SIDE_FIRST_LAYER][BLOCK_SIDE_FIRST_LAYER+1]; Non possiamo ancora giustificare il miglioramento nei tempi.
		__shared__ DATA shared_w[BLOCK_SIDE_FIRST_LAYER][BLOCK_SIDE_FIRST_LAYER];
		__shared__ DATA shared_h2h[BLOCK_SIDE_FIRST_LAYER][BLOCK_SIDE_FIRST_LAYER];

		int t_index_w = wid + tx + ty*col_w;
		int t_index_h2h = h2h_id + tx + ty*row_w;

		//Attenzione alla divergenza dei threads (vedi CCC pag.137)
		shared_h2h[ty][tx] = (t_index_h2h < num_pattern*row_w) ? (h2h[t_index_h2h]) : (0.0f);
		shared_w[ty][tx] = (t_index_w < col_w*row_w) ? (w[t_index_w]) : (0.0f);

		__syncthreads();

		current_inc = row_w - (block_r_border - block_dim);

		min = (current_inc < block_dim) ? (current_inc) : (block_dim);

		for (int k = 0; k < min; k++) {
			partial += shared_h2h[ty][k] * shared_w[k][tx];
		}

		__syncthreads();
	}

	//Attenzione alla divergenza dei threads (vedi CCC pag.137)
	if (dest_x < col_w && dest_y < num_pattern) {
		h2h_dest[dest_y*col_w + dest_x] = (DATA)1.0 / (DATA)(1.0 + exp(-(partial + biases[dest_x]))); //SIGMA
	}
}

/*patt_per_step � il numero di pattern (quando possibile...) da considerare in ciascuna iterazione su h2h*/
/*Questo kernel ad ogni passo incrementa il puntatore ad h2h di num_patt_per_step*NEURO_L_L_1 (e similmente h2h_dest),
controlla che sia ancora nel range di h2h, e calcola num_pattern (vedi sopra) in funzione dei
pattern mancanti*/
//Dove ora c'� STREAMSIZE prima c'era TOTAL_PATT
__global__ void MMMulDev(DATA *h2h, DATA *w, DATA *biases, DATA *h2h_dest, unsigned int row_w, unsigned int col_w, unsigned int patt_per_step) {

	unsigned int current_patts;
	unsigned int remaining_patts;
	const int pos_block_y = blockIdx.y*blockDim.x; //Posizione del blocco corrente rispetto alla griglia lungo le y
												   //Assumiamo che i blocchi siano quadrati (blockDim.x = blockDim.y)			
	for (unsigned int x = 0; x < STREAMSIZE; x += patt_per_step) {

		remaining_patts = STREAMSIZE - x;
		current_patts = (remaining_patts < patt_per_step) ? (remaining_patts) : (patt_per_step);

		if (pos_block_y >= current_patts) { return; }

		MMMulDevPartial(h2h + x*row_w, w, biases, h2h_dest + x*col_w, row_w, col_w, current_patts);
	}
}


/*HOST*/

/*FIRT PHASE OF THE ALGORITHM -- THE INPUT IS TRANSMITTED VIA THE NETWORK*/
void feedforward(DATA *INPUT, DATA **WeightH2H, DATA **BiasH2H, DATA **dev_H2H, DATA **dev_WeightH2H, DATA **dev_BIASH2H, int *nupl, int layers) {

	hipStream_t streams[NSTREAMS];
	for (int i = 0; i < NSTREAMS; i++) {
		HANDLE_CUDA(hipStreamCreate(&streams[i]));
	}

	//Grid setting
	dim3 grid, block;
	unsigned int patt_per_step;

	int offset;

	//Compattare le copie comuni a tutti gli streams (delle matrici dei pesi e dei bias) con una struct?...in attesa dell'Ebreo!
	for (int l = 0; l < (layers - 1); l++) {

		HANDLE_CUDA(hipMemcpy(dev_WeightH2H[l], WeightH2H[l], sizeof(DATA)*nupl[l] * nupl[l + 1], hipMemcpyHostToDevice));
		HANDLE_CUDA(hipMemcpy(dev_BIASH2H[l], BiasH2H[l], sizeof(DATA)*nupl[l + 1], hipMemcpyHostToDevice));
	}

	for (int i = 0; i < NSTREAMS; i++) {

		block.x = gs.block[0];
		block.y = gs.block[0];
		grid.x = (nupl[1] + block.x - 1) / block.x;
		grid.y = gs.grid[0] / grid.x;

		patt_per_step = grid.y * block.y;

		offset = i*STREAMSIZE;
		HANDLE_CUDA(hipMemcpyAsync(dev_H2H[0] + offset*nupl[0], INPUT + offset*nupl[0], nupl[0] * STREAMSIZE * sizeof(DATA), hipMemcpyHostToDevice, streams[i]));
		MMMulDev << <grid, block, 0, streams[i] >> > (dev_H2H[0] + offset*nupl[0], dev_WeightH2H[0], dev_BIASH2H[0], dev_H2H[1] + offset*nupl[1], nupl[0], nupl[1], patt_per_step);
		
		for (int l = 1; l < (layers - 1); l++) {

			block.x = gs.block[l];
			block.y = gs.block[l];
			grid.x = (nupl[l + 1] + block.x - 1) / block.x;
			grid.y = gs.grid[l] / grid.x;

			patt_per_step = grid.y * block.y;


			MMMulDev << <grid, block, 0, streams[i] >> > (dev_H2H[l] + offset*nupl[l], dev_WeightH2H[l], dev_BIASH2H[l], dev_H2H[l + 1] + offset*nupl[l + 1], nupl[l], nupl[l + 1], patt_per_step);
		}
	}
}

/*UTILITY FUNCTIONS*/

void HOST_feedforward(DATA *INPUT, DATA **W, DATA **BIAS, DATA **H2H, int *nupl) {

	MMMulHost(INPUT, W[0], BIAS[0], H2H[0], TOTAL_PATT, nupl[0], nupl[1]);
	MMMulHost(H2H[0], W[1], BIAS[1], H2H[1], TOTAL_PATT, nupl[1], nupl[2]);
	MMMulHost(H2H[1], W[2], BIAS[2], H2H[2], TOTAL_PATT, nupl[2], nupl[3]);

}

/*Print a matrix*/
void printMat(DATA *mat, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		printf("ROW %d : {", i);
		for (int j = 0; j < cols; j++) {
			printf("%f - ", mat[i*cols + j]);
		}
		printf("}");
		printf("\n\n");
	}
	printf("\n\n");
}

/*On host multiplication*/
void MMMulHost(DATA *H2H, DATA *W, DATA *BIAS, DATA *H2H_RES, int row_H2H, int col_H2H, int col_W) {

	for (int i = 0; i < row_H2H; i++) {
		for (int j = 0; j < col_W; j++) {
			DATA prod = 0.0;
			for (int k = 0; k < col_H2H; k++) {
				prod += H2H[i*col_H2H + k] * W[k*col_W + j];
			}
			H2H_RES[i*col_W + j] = (DATA)1.0 / (DATA)(1.0 + exp(-(prod + BIAS[j]))); // bias added
		}
	}
}

/*Check device*/
BOOL matsAreEquals(DATA *A, DATA *B, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) { // the first column is for adapting the data
			float err = fabs(A[i*cols + j] - B[i*cols + j]);
			//printf("Error in i=%d,j=%d: %f\n", i, j, err);
			if (err >= MAX_ERR) { printf("row: %d, col: %d\n", i, j); return 0; }
		}
	}
	return 1;
}

/*ALLOCATION FUNCTIONS*/

/*Allocation and initialization of host memory*/
void HOST_alloc_init(DATA** H2H_MAT, DATA** W_MAT, DATA** BIAS_MAT, int* nupl) {

	//Bisogner� inserire i controlli sulle malloc
	for (int layer = 0; layer<(TOTAL_LAYER - 1); layer++) {
		W_MAT[layer] = (DATA*)malloc(nupl[layer] * nupl[layer + 1] * sizeof(DATA));
		BIAS_MAT[layer] = (DATA*)malloc(nupl[layer + 1] * sizeof(DATA));
		H2H_MAT[layer] = (DATA*)malloc(nupl[layer + 1] * TOTAL_PATT * sizeof(DATA));

		for (int i = 0; i < nupl[layer]; i++) {
			for (int j = 0; j < nupl[layer + 1]; j++) {
				W_MAT[layer][i*nupl[layer + 1] + j] = (DATA)rand() / (DATA)RAND_MAX;
				BIAS_MAT[layer][j] = (DATA)rand() / (DATA)RAND_MAX;
			}
		}
	}
}

/*Allocation of device memory (by host)*/
void CUDA_alloc(DATA** DEV_H2H_MAT, DATA** DEV_W_MAT, DATA** DEV_BIAS_MAT, int* nupl) {

	for (int layer = 0; layer<(TOTAL_LAYER - 1); layer++) {
		HANDLE_CUDA(hipMalloc(&(DEV_H2H_MAT[layer]), nupl[layer] * TOTAL_PATT * sizeof(DATA)));
		HANDLE_CUDA(hipMalloc(&(DEV_W_MAT[layer]), nupl[layer] * nupl[layer + 1] * sizeof(DATA)));
		HANDLE_CUDA(hipMalloc(&(DEV_BIAS_MAT[layer]), nupl[layer + 1] * sizeof(DATA)));
	}
	HANDLE_CUDA(hipMalloc(&(DEV_H2H_MAT[TOTAL_LAYER - 1]), nupl[TOTAL_LAYER - 1] * TOTAL_PATT * sizeof(DATA)));
}

/*DEALLOCATION FUNCTIONS*/

/*Deallocation of host memory*/
void HOST_dealloc(DATA* INPUT, DATA* TARGET, DATA** H2H_MAT, DATA** W_MAT, DATA** BIAS_MAT) {

	for (int layer = 0; layer<(TOTAL_LAYER - 1); layer++) {
		free(H2H_MAT[layer]);
		free(W_MAT[layer]);
		free(BIAS_MAT[layer]);
	}

	hipHostFree(INPUT); //pinned memory free
	free(TARGET);
	free(H2H_MAT);
	free(W_MAT);
	free(BIAS_MAT);
}

/*Deallocation of device memory (called by host)*/
void CUDA_dealloc(DATA** DEV_H2H_MAT, DATA** DEV_W_MAT, DATA** DEV_BIAS_MAT) {
	for (int layer = 0; layer<(TOTAL_LAYER - 1); layer++) {
		hipFree(DEV_H2H_MAT[layer]);
		hipFree(DEV_W_MAT[layer]);
		hipFree(DEV_BIAS_MAT[layer]);
	}
	hipFree(DEV_H2H_MAT[TOTAL_LAYER - 1]);
}

//NON CANCELLARE !!! INSERIRE NEL MAIN PER FARE TEST DI CORRETTEZZA !!!

//hipMemcpy(OUTPUT_MAT, DEV_H2H_MAT[TOTAL_LAYER - 1], NEURO_OUTPUT*TOTAL_PATT * sizeof(DATA), hipMemcpyDeviceToHost);

//printMat(OUTPUT_MAT, TOTAL_PATT, NEURO_OUTPUT);

//HOST_feedforward(INPUT_MAT, W_MAT, BIAS_MAT, H2H_MAT, nupl);
//printMat(H2H_MAT[2], TOTAL_PATT, NEURO_OUTPUT);

//BOOL b = matsAreEquals(OUTPUT_MAT, H2H_MAT[2], TOTAL_PATT, NEURO_OUTPUT);
//printf("%d\n", b);
